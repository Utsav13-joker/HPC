#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

__device__ int is_a_match(char *no_of_tries) {
  char password1[] = "AV7201";
  char password2[] = "FT2702";
  char password3[] = "IR7503";
  char password4[] = "SC5504";

  char *p = no_of_tries;
  char *q = no_of_tries;
  char *r = no_of_tries;
  char *s = no_of_tries;
  char *pw1 = password1;
  char *pw2 = password2;
  char *pw3 = password3;
  char *pw4 = password4;

  while(*p == *pw1) {
   if(*p == '\0')
    {
    printf("Found password: %s\n",password1);
      break;
    }

    p++;
    pw1++;
  }
    
  while(*q == *pw2) {
   if(*q == '\0')
    {
    printf("Found password: %s\n",password2);
      break;
}

    q++;
    pw2++;
  }

  while(*r == *pw3) {
   if(*r == '\0')
    {
    printf("Found password: %s\n",password3);
      break;
    }

    r++;
    pw3++;
  }

  while(*s == *pw4) {
   if(*s == '\0')
    {
    printf("Found password: %s\n",password4);
      return 1;
    }

    s++;
    pw4++;
  }
  return 0;

}


__global__ void  kernel() {
char a,b,c,d;
 
  char password[7];
  password[6] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
    
password[0] = firstValue;
password[1] = secondValue;
    for(a='0'; a<='9'; a++){
      for(b='0'; b<='9'; b++){
        for(c='0';c<='9';c++){
          for(d='0';d<='9';d++){
            password[2] = a;
            password[3] = b;
            password[4]= c;
            password[5]=d;
          if(is_a_match(password)) {
        //printf("Success");
          }
             else {
         //printf("tried: %s\n", password);          
            }
          }
        } 
      }
   }
}
int time_variance(struct timespec *start,
                    struct timespec *end,
                    long long int *variance) {
  long long int sec =  end->tv_sec - start->tv_sec;
  long long int nsec =  end->tv_nsec - start->tv_nsec;

  if(nsec < 0 ) {
    sec--;
    nsec += 1000000000;
  }
  *variance = sec * 1000000000 + nsec;
  return !(*variance > 0);
}


int main() {

  struct  timespec start, end;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &end);
  time_variance(&start, &end, &time_elapsed);
  printf("Time elapsed was %lldnsecs or %0.9lfs\n", time_elapsed, 
         (time_elapsed/1.0e9)); 
  return 0;
}



