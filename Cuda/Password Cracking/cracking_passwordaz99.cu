#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

__device__ int is_a_match(char *no_of_tries) {
  char password1[] = "AV72";
  char password2[] = "FT27";
  char password3[] = "IR75";
  char password4[] = "SC55";

  char *p = no_of_tries;
  char *q = no_of_tries;
  char *r = no_of_tries;
  char *s = no_of_tries;
  char *pw1 = password1;
  char *pw2 = password2;
  char *pw3 = password3;
  char *pw4 = password4;

  while(*p == *pw1) {
   if(*p == '\0')
    {
    printf("Found password: %s\n",password1);
      break;
    }

    p++;
    pw1++;
  }
    
  while(*q == *pw2) {
   if(*q == '\0')
    {
    printf("Found password: %s\n",password2);
      break;
}

    q++;
    pw2++;
  }

  while(*r == *pw3) {
   if(*r == '\0')
    {
    printf("Found password: %s\n",password3);
      break;
    }

    r++;
    pw3++;
  }

  while(*s == *pw4) {
   if(*s == '\0')
    {
    printf("Found password: %s\n",password4);
      return 1;
    }

    s++;
    pw4++;
  }
  return 0;
}


__global__ void  kernel() {
char m,n;
 
  char password[5];
  password[4] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
    
password[0] = firstValue;
password[1] = secondValue;
    for(m='0'; m<='9'; m++){
      for(n='0'; n<='9'; n++){
            password[2] = m;
            password[3] = n;
          if(is_a_match(password)) {
        //printf(  "Password cracking done");
          }
             else {
         //printf("Failed: %s\n", password);          
            }
          }
        } 
      
}
int time_variation(struct timespec *start, struct timespec *end, 
                    long long int *variation) {
  long long int dsec =  end->tv_sec - start->tv_sec; 
  long long int dnsec =  end->tv_nsec - start->tv_nsec; 

  if(dnsec < 0 ) {
    dsec--;
    dnsec += 1000000000; 
  } 
  *variation = dsec * 1000000000 + dnsec;
  return !(*variation > 0);
}


int main() {

  struct timespec start, end;   
  long long int time_elapsed;

  clock_gettime(CLOCK_MONOTONIC, &start);

	kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &end);
  time_variation(&start, &end, &time_elapsed);
  printf("Time elapsed was %lldnsecs or %0.9lfs\n", time_elapsed,(time_elapsed/1.0e9)); 
  return 0;
}




